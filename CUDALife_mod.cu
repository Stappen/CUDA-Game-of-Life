#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
// CUDA includes
#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""
#include <hip/device_functions.h>

const int ARRAY_SIZE = 30;

__device__ int getIndex(int x, int y)
{
	return ((y * ARRAY_SIZE) + x);
}

__device__ int mod(int num, int mod)
{
	int ret = a % b;
	if(ret < 0)
		ret+=b;
	return ret;
}

__device__ int isCellAlive(int *d_read)
{
	int up, down, left, right;
	int sm1 = ARRAY_SIZE - 1;		// size-1
	
	up = mod(threadIdx.y - 1, sm1);
	down = mod(threadIdx.y + 1, sm1);
	left = mod(threadIdx.x - 1, sm1);
	right = mod(threadIdx.x + 1, sm1);

	int count = d_read[getIndex(left, up)] + d_read[getIndex(threadIdx.x, up)] + d_read[getIndex(right, up)] + d_read[getIndex(left, threadIdx.y)] + d_read[getIndex(right, threadIdx.y)] + d_read[getIndex(left, down)] + d_read[getIndex(threadIdx.x, down)] + d_read[getIndex(right, down)];

	// check rules of the game
	int rule1,rule2;
	rule1 = ((count == 2) || (count == 3)) && (d_read[getIndex(threadIdx.x, threadIdx.y)]);		// count == 2 or 3 && the current cell == 1
	rule2 = (count == 3) && (d_read[getIndex(threadIdx.x, threadIdx.y)] == 0);					// count == 3 && the current cell == 0

	return (rule1 || rule2);
}

__global__ void simulate(int *d_read, int *d_write)
{
	int i = getIndex(threadIdx.x, threadIdx.y);

	d_write[i] = isCellAlive(d_read);

	// swap values then reset the write buffer
	d_read[i] = d_write[i];
	d_write[i] = 0;
}

void printGrid(int grid[ARRAY_SIZE][ARRAY_SIZE])
{
	for (int y = 0; y < ARRAY_SIZE; y++){
		for (int x = 0; x < ARRAY_SIZE; x++)
		{
			if (grid[x][y] == 1)
			{
				std::cout << " #";
			}
			else
			{
				std::cout << " .";
			}
		}
		std::cout << std::endl;
	}
	std::cout << "\n\n";
}

int main()
{
	// create the host grid used for printing
	int h_read[ARRAY_SIZE][ARRAY_SIZE] = { 0 };

	// glider for testing
	h_read[3][2] = 1;
	h_read[4][3] = 1;
	h_read[2][4] = 1;
	h_read[3][4] = 1;
	h_read[4][4] = 1;

	printGrid(h_read);

	// declate GPU grid pointers
	int *d_read;
	int *d_write;
	// allocate memory in the device's memory space
	hipMalloc(&d_read, sizeof(int) * (ARRAY_SIZE * ARRAY_SIZE));
	hipMalloc(&d_write, sizeof(int) * (ARRAY_SIZE * ARRAY_SIZE));

	// copy the input data from the host's memory space to the device's memory space
	hipMemcpy(d_read, h_read, (sizeof(int) * (ARRAY_SIZE * ARRAY_SIZE)), hipMemcpyHostToDevice);
	memset(h_read, 0, sizeof(h_read));
	hipMemcpy(d_write, h_read, (sizeof(int) * (ARRAY_SIZE * ARRAY_SIZE)), hipMemcpyHostToDevice);

	// run the kernel
	int gen = 10;

	for (int _gen = 0; _gen < gen; _gen++)
	{
		simulate << <1, dim3(ARRAY_SIZE, ARRAY_SIZE) >> >(d_read, d_write);
	}

	// copy the input data from the device's memory space to the host's memory space
	hipMemcpy(h_read, d_read, (sizeof(int) * (ARRAY_SIZE * ARRAY_SIZE)), hipMemcpyDeviceToHost);

	printGrid(h_read);

	hipFree(d_read);
	hipFree(d_write);
	//std::cin.get();		// stop console from automatically closing (for testing)
	return 0;
}
